
#include <hip/hip_runtime.h>
__global__ void MatAdd(float* c,
                            const float* a,
                            const float* b,
                            int n)
{
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    int j = blockIdx.y * blockDim.y + threadIdx.y;
    int idx = j*n + i;
    if (i < n && j < n)
        c[idx] = a[idx] + b[idx];
}

void launch_add2(float* c,
                 const float* a,
                 const float* b,
                 int n) {
    dim3 block(16, 16);
    dim3 grid(n/block.x, n/block.y);

    MatAdd<<<grid, block>>>(c, a, b, n);
}